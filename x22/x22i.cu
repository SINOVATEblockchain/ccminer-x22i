#include "hip/hip_runtime.h"
/**
 * X22I algorithm
 */

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"

#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"

#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"

#include "sph/sph_tiger.h"
#include "lyra2/Lyra2.h"
}
#include "sph/sph_streebog.h"
#include "SWIFFTX/SWIFFTX.h"

#include "miner.h"
#include "cuda_helper.h"
#include "x11/cuda_x11.h"

static uint32_t *d_hash[MAX_GPUS], *d_hash1[MAX_GPUS], *d_hash2[MAX_GPUS], *d_hash3[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

//extern void x16_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);

extern void x13_hamsi512_cpu_init(int thr_id, uint32_t threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x13_fugue512_cpu_free(int thr_id);

extern void x14_shabal512_cpu_init(int thr_id, uint32_t threads);
extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int flag);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x15_whirlpool_cpu_free(int thr_id);

extern void x17_sha512_cpu_init(int thr_id, uint32_t threads);
extern void x17_sha512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x17_haval256_cpu_init(int thr_id, uint32_t threads);
extern void x17_haval256_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const int outlen);

extern void streebog_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);

extern void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, int order);

extern void tiger192_cpu_hash_64(int thr_id, int threads, uint32_t *d_hash);
extern void sha256_cpu_hash_64(int thr_id, int threads, uint32_t *d_hash);

extern void h_InitializeSWIFFTX();
extern void swifftx512_cpu_hash_64(int thr_id, int threads, uint32_t *d_hash, uint32_t *d_hash1, uint32_t *d_hash2, uint32_t *d_hash3);

extern void x11_echo512_cpu_init(int thr_id, uint32_t throughput);
extern void x11_echo512_cpu_hash_64_sp(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x11_shavite512_cpu_hash_64_sp(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x13_fugue512_cpu_hash_64_sp(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x13_hamsi512_cpu_hash_64_sp(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x14_shabal512_cpu_hash_64_sp(int thr_id, uint32_t threads, uint32_t *d_hash);


// X22I CPU Hash (Validation)
extern "C" void x22ihash(void *output, const void *input)
{
	//unsigned char _ALIGN(64) hash[128];
	unsigned char hash[64 * 4] = {0}, hash2[64] = {0};

	// x11 + hamsi12-fugue13-shabal14-whirlpool15-sha512-haval256

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;
	sph_whirlpool_context ctx_whirlpool;
	sph_sha512_context ctx_sha512;
	sph_haval256_5_context ctx_haval;
	sph_tiger_context         ctx_tiger;
	sph_gost512_context       ctx_gost;
	sph_sha256_context        ctx_sha;

	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hash);

	sph_bmw512_init(&ctx_bmw);
	/*
	// ZERO hash test, leads to "624381675728598999"
	unsigned char test[64] = {0};
	sph_bmw512(&ctx_bmw, (const void*) test, 64);
	*/
	sph_bmw512(&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, (const void*) hash, 64);
	sph_luffa512_close (&ctx_luffa, hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, hash);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, (const void*) hash, 64);
	sph_hamsi512_close(&ctx_hamsi, hash);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, (const void*) hash, 64);
	sph_fugue512_close(&ctx_fugue, hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, (const void*) hash, 64);
	sph_shabal512_close(&ctx_shabal, &hash[64]);

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool (&ctx_whirlpool, (const void*) &hash[64], 64);
	sph_whirlpool_close(&ctx_whirlpool, &hash[128]);

	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512,(const void*) &hash[128], 64);
	sph_sha512_close(&ctx_sha512,(void*) &hash[192]);

	InitializeSWIFFTX();
	ComputeSingleSWIFFTX((unsigned char*)hash, (unsigned char*)hash2, false);

	memset(hash, 0, 64);
	sph_haval256_5_init(&ctx_haval);
	sph_haval256_5(&ctx_haval,(const void*) hash2, 64);
	sph_haval256_5_close(&ctx_haval,hash);

	memset(hash2, 0, 64);
	sph_tiger_init(&ctx_tiger);
	sph_tiger (&ctx_tiger, (const void*) hash, 64);
	sph_tiger_close(&ctx_tiger, (void*) hash2);

	memset(hash, 0, 64);
	LYRA2((void*) hash, 32, (const void*) hash2, 32, (const void*) hash2, 32, 1, 4, 4);

	sph_gost512_init(&ctx_gost);
	sph_gost512 (&ctx_gost, (const void*) hash, 64);
	sph_gost512_close(&ctx_gost, (void*) hash);

	sph_sha256_init(&ctx_sha);
	sph_sha256 (&ctx_sha, (const void*) hash, 64);
	sph_sha256_close(&ctx_sha, (void*) hash);

	/*
  // zero hash test print
  printf("%lu\n", ((uint64_t*)(hash))[0]);
  */

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_x22i(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 19); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	uint64_t gpu_ram_size = 16 * sizeof(uint32_t) * throughput;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x08ff;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		size_t matrix_sz = 16 * sizeof(uint64_t) * 4 * 3;
		// SM 3 implentation requires a bit more memory
		if (device_sm[dev_id] < 500 || cuda_arch[dev_id] < 500) matrix_sz = 16 * sizeof(uint64_t) * 4 * 4;
		CUDA_CALL_OR_RET_X(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput), -1);

		cuda_get_arch(thr_id);
		x11_echo512_cpu_init(thr_id, throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffaCubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x14_shabal512_cpu_init(thr_id, throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 0);
		x17_sha512_cpu_init(thr_id, throughput);
		h_InitializeSWIFFTX();
		x17_haval256_cpu_init(thr_id, throughput);
		lyra2v2_cpu_init(thr_id, throughput, d_matrix[thr_id]);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash [thr_id], gpu_ram_size), 0);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash1[thr_id], gpu_ram_size), 0);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash2[thr_id], gpu_ram_size), 0);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash3[thr_id], gpu_ram_size), 0);

		cuda_check_cpu_init(thr_id, throughput);

		api_set_throughput(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t _ALIGN(64) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	int warn = 0;

	do {
		int order = 0;

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;

		/*
		// zero hash test
		hipMemset(d_hash[thr_id], 0, gpu_ram_size);
		*/

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		
		//x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64_sp(thr_id, throughput, d_hash[thr_id]); order++;

		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		//x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64_sp(thr_id, throughput, d_hash[thr_id]); order++;

		//x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_hamsi512_cpu_hash_64_sp(thr_id, throughput, d_hash[thr_id]); order++;

		//x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64_sp(thr_id, throughput, d_hash[thr_id]); order++;

		hipMemcpy(d_hash1[thr_id], d_hash[thr_id], gpu_ram_size, hipMemcpyDeviceToDevice);
		//x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash1[thr_id], order++);
		x14_shabal512_cpu_hash_64_sp(thr_id, throughput, d_hash1[thr_id]); order++;

		hipMemcpy(d_hash2[thr_id], d_hash1[thr_id], gpu_ram_size, hipMemcpyDeviceToDevice);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash2[thr_id], order++);

		hipMemcpy(d_hash3[thr_id], d_hash2[thr_id], gpu_ram_size, hipMemcpyDeviceToDevice);
		x17_sha512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash3[thr_id]); order++;

		swifftx512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], d_hash1[thr_id], d_hash2[thr_id], d_hash3[thr_id]);

		x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], 512); order++;
		tiger192_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);

		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], (uint64_t*) d_hash[thr_id], order++);		// add 0 padding????

		streebog_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		sha256_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);

		/*
		// zero hash test print
		uint64_t tmp;
		hipMemcpy(&tmp, d_hash[thr_id], 8, hipMemcpyDeviceToHost);
		printf("D: %lu\n", tmp);
		*/

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			x22ihash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					x22ihash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				// x11+ coins could do some random error, but not on retry
				gpu_increment_reject(thr_id);
				if (!warn) {
					warn++;
					pdata[19] = work->nonces[0] + 1;
					continue;
				} else {
					if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
					warn = 0;
				}
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_x22i(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_matrix[thr_id]);
	hipFree(d_hash [thr_id]);
	hipFree(d_hash1[thr_id]);
	hipFree(d_hash2[thr_id]);
	hipFree(d_hash3[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);
	x13_fugue512_cpu_free(thr_id);
	x15_whirlpool_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
